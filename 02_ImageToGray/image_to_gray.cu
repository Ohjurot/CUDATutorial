#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>

#include "stb_image.h"
#include "stb_image_write.h"

struct Pixel
{
    unsigned char r, g, b, a;
};

void ConvertImageToGrayCpu(unsigned char* imageRGBA, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;
        }
    }
}

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}

int main(int argc, char** argv)
{
    // Check argument count
    if (argc < 2)
    {
        std::cout << "Usage: 02_ImageToGray <filename>";
        return -1;
    }

    // Open image
    int width, height, componentCount;
    std::cout << "Loading png file...";
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        std::cout << std::endl << "Failed to open \"" << argv[1] << "\"";
        return -1;
    }
    std::cout << " DONE" << std::endl;

    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        std::cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    /*
    // Process image on cpu
    std::cout << "Processing image...";
    ConvertImageToGrayCpu(imageData, width, height);
    std::cout << " DONE" << std::endl;
    */

    // Copy data to the gpu
    std::cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
    std::cout << " DONE" << std::endl;

    // Process image on gpu
    std::cout << "Running CUDA Kernel...";
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu<<<gridSize, blockSize>>>(ptrImageDataGpu);
    auto err = hipGetLastError();
    std::cout << " DONE" << std::endl; 

    // Copy data from the gpu
    std::cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    std::cout << " DONE" << std::endl;

    // Build output filename
    std::string fileNameOut = argv[1];
    fileNameOut = fileNameOut.substr(0, fileNameOut.find_last_of('.')) + "_gray.png";

    // Write image back to disk
    std::cout << "Writing png to disk...";
    stbi_write_png(fileNameOut.c_str(), width, height, 4, imageData, 4 * width);
    std::cout << " DONE";

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
}
